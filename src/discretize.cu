#include "discretize.h"
#include <hip/hip_runtime.h>
#include <map>
#include <vector>
#include <moab/ErrorCode>

#define CUDA_CHECK(value, label) {              \
   hipError_t c = (value);                     \
   if (c != hipSuccess) {                      \
   fprintf(stderr,                              \
     "Error: '%s' at line %d in %s\n",          \
     hipGetErrorString(c),__LINE__,__FILE__);  \
   goto label;                                  \
   } }

std::vector<double> cuda_rayfire(moab::Instance MBI, moab::GeomTopoTool GTT,
                                 moab::GeomQueryTool GQT, mesh_row row,
                                 std::vector<moab::EntityHandle> vol_handles) {

  std::vector<std::map<int, std::vector<double> > > row_totals;
  std::vector<double> width;
  moab::ErrorCode rval; 
  for (int i = 0; i < row.d3divs.size() - 1; i++) {
    width.push_back(row.d3divs[i+1] - row.d3divs[i]);
  }

  row_totals.resize(width.size());
  moab::EntityHandle root;
  ErrorCode rval = geomTopoTool->get_root(vol_handles[0], root);
// somehow we need to get the OBBTree into an array of doubles.
// We'll call it obbs[]
// Unfortunately, we'll need to hope that the tree is balanced well, because
// we'll only be able to find things by having them ordered; root, level 1,
// level 1, level 2 ...
// Any leaves that aren't there will be left as gaps in the array.
// Each OBB will be taken up by
// center[0], center[1], center[2], length[0], length[1], length[2],
// axes[0], ..., axis[8], radius
  double obbs[size_of_OBBT] = {};
  // The size of this one is tricky; we don't know how many we'll end up hitting
  // so we give it enough space for each of the leaf entitites in the OBBT plus
  // each mesh boundary, assuming only two triangles per OBB.
  double distances[size_of_OBBT/16 + width.size()];
  // Oh shoot, that's how many we need PER RAY! and it's useless if we don't
  // fire a bunch of rays, as that's where the parallelization happens. There
  // is REALLY no memory for this.
  double *d_obbs, *d_distances, *d_width;

  CUDA_CHECK(hipMalloc(d_obbs, size_of_OBBT*sizeof(double)), cuda_error)
  CUDA_CHECK(hipMalloc(d_width, width.size()*sizeof(double)), cuda_error)
  CUDA_CHECK(hipMalloc(d_distances, (size_of_OBBT/16 + width.size())*sizeof(double)), cuda_error)
  
  CUDA_CHECK(hipMemcpy(d_obbs, obbs, size_of_OBBT*sizeof(double), hipMemcpyHostToDevice), cuda_error)
  CUDA_CHECK(hipMemcpy(d_width, &width[0], width.size()*sizeof(double), hipMemcpyHostToDevice), cuda_error)
  
  N = row.num_rays;
  // We're not using shared memory, so we're going to go with the maximum
  // number of threads per block.
  double blocksPerGrid = (N+1023)/1024;
  double threadsPerBlock = 1024;
  // There also needs to be a way to get the starting points of each ray;
  // didn't get that far.
  bogus_kernel<<<blocksPerGrid,threadsperblock>>>(d_obbs,d_width,d_distances,N,row.grid);

  // Don't actually do this. This would mangle the data.
  CUDA_CHECK(hipMemcpy(distances, d_distances, (size_of_OBBT/16 + width.size())*sizeof(double), hipMemcpyDeviceToHost), cuda_error)
  std::vector<int> distances_vec(std::begin(distances), std::end(distances));
  hipFree(d_distances);
  hipFree(d_obbs);
  hipFree(d_width);
  
  return distances_vec
}
